#include "hip/hip_runtime.h"
#include <cinttypes>
#include <cstdint>
#include <cstdlib>
#include <cstdio>
#include <cstring>
#include <cmath>
#include <algorithm>
#include <chrono>
#include <map>
#include <hip/hip_runtime.h>

using namespace std::chrono;

#define DEFAULT_SIEVE_SIZE 1'000'000

// The actual number of threads used can be lower, if there aren't enough "chunks of work" to keep this
//   number of threads busy.
#define MAX_THREADS 256

// Highest prime number for which we'll use a rolling bit mask. This value cannot be higher than the
//   number of bits per word! Set to 0 to disable the use of a rolling bit mask. The relevant code won't
//   be compiled then, thus reducing performance side effects to zero. 
#define ROLLING_LIMIT 29

// If defined, the code will show debug output and run both unmarking methods once.
//#define DEBUG

// Modify the following 4 lines if you want to switch to another word type for the sieve buffer. Two sets
//   of values are supported:
//   - for 32 bit words: 
//     sieve_t = unsigned int, BITS_PER_WORD = UINT32_WIDTH, MAX_WORD_VALUE = UINT32_MAX, WORD_SHIFT = 5
//   - for 64 bit words: 
//     sieve_t = unsigned long long, BITS_PER_WORD = UINT64_WIDTH, MAX_WORD_VALUE = UINT64_MAX, WORD_SHIFT = 6
typedef unsigned int sieve_t;
#define BITS_PER_WORD UINT32_WIDTH
#define MAX_WORD_VALUE UINT32_MAX
#define WORD_SHIFT 5

// The following defines are derived from the previous 4, so it's not necessary to modify these
#define BYTES_PER_WORD (BITS_PER_WORD >> 3)
#define MAX_BIT_INDEX (BITS_PER_WORD - 1)
#define WORD_INDEX(index) (index >> WORD_SHIFT)
#define BIT_INDEX(index) (index & MAX_BIT_INDEX)
// This is actually BITS_PER_WORD * 2 - 1, but this is a "cheap" way to get there
#define SIEVE_WORD_MASK ~uint64_t(BITS_PER_WORD + MAX_BIT_INDEX)

#if ROLLING_LIMIT > BITS_PER_WORD
  #error "ROLLING_LIMIT can't be greater than the number of bits per word!"
#endif

__global__ void initialize_buffer(uint64_t blockSize, uint64_t wordCount, sieve_t *sieve)
{
    const uint64_t startIndex = uint64_t(blockIdx.x) * blockSize;
    const uint64_t endIndex = ullmin(startIndex + blockSize, wordCount);

    for (uint64_t index = startIndex; index < endIndex; index++)
        sieve[index] = MAX_WORD_VALUE;
}

__global__ void unmark_multiples_threads(uint32_t primeCount, uint32_t *primes, uint64_t halfSize, uint32_t sizeSqrt, sieve_t *sieve)
{
    // We unmark every "MAX_THREADS"th prime's multiples, starting with our thread index
    for (uint32_t primeIndex = threadIdx.x; primeIndex < primeCount; primeIndex += MAX_THREADS) 
    {
        const uint32_t prime = primes[primeIndex];
        const uint64_t primeSquared = uint64_t(prime) * prime;

        // Unmark multiples starting at just beyond the square root of the sieve size or the square of the prime, 
        //   whichever is larger.
        uint64_t firstUnmarked = primeSquared > sizeSqrt ? primeSquared : ((sizeSqrt / prime + 1) * prime);
        // We're marking off odd multiples only, so make sure we start with one of those!
        if (!(firstUnmarked & 1))
            firstUnmarked += prime;

        for (uint64_t index = firstUnmarked >> 1; index <= halfSize; index += prime) 
            // Clear the bit in the word that corresponds to the last part of the index 
            atomicAnd(&sieve[WORD_INDEX(index)], ~(sieve_t(1) << BIT_INDEX(index)));
    }
}

__global__ void unmark_multiples_blocks(uint32_t primeCount, uint32_t *primes, uint64_t halfSize, uint32_t sizeSqrt, uint32_t maxBlockIndex, uint64_t blockSize, sieve_t *sieve)
{
    // Calculate the start and end of the block we need to work on, at buffer word boundaries. 
    //   Note that the first variable is a number in sieve space...
    uint64_t blockStart = uint64_t(blockIdx.x) * blockSize + sizeSqrt;
    //   ...and the second is an index in the sieve buffer (representing odd numbers only)
    const uint64_t lastIndex = (blockIdx.x == maxBlockIndex) ? halfSize : (((blockStart + blockSize) & SIEVE_WORD_MASK) >> 1) - 1;

    // If this is not the first block, we actually start at the beginning of the first block word
    if (blockIdx.x != 0)
        blockStart &= SIEVE_WORD_MASK;

#ifdef DEBUG
    printf("  - block %d: blockStart = %" PRIu64 " (index %" PRIu64 "), lastIndex = %" PRIu64 ".\n", blockIdx.x, blockStart, (blockStart >> 1), lastIndex);
#endif

    for (uint32_t primeIndex = 0; primeIndex < primeCount; primeIndex++)
    {
        const uint32_t prime = primes[primeIndex];
        const uint64_t primeSquared = uint64_t(prime) * prime;

        // Unmark multiples starting at just beyond the start of our block or the square of the prime, 
        //   whichever is larger.
        uint64_t firstUnmarked = primeSquared >= blockStart ? primeSquared : ((blockStart / prime + 1) * prime);
        // We're marking off odd multiples only, so make sure we start with one of those!
        if (!(firstUnmarked & 1))
            firstUnmarked += prime;

    #if ROLLING_LIMIT > 0
        if (prime <= ROLLING_LIMIT)
        {
            uint64_t index = firstUnmarked >> 1;
            if (index > lastIndex)
                continue;

            uint64_t wordIndex = WORD_INDEX(index);
            uint32_t bitIndex = BIT_INDEX(index);                
            sieve_t bitMask = 0;

            do
            {
                if (bitIndex > MAX_BIT_INDEX) 
                {
                    // Clear the bits that are set in the mask
                    sieve[wordIndex++] &= ~bitMask;
                    bitIndex %= BITS_PER_WORD;
                    bitMask = sieve_t(1) << bitIndex;
                }
                else
                    bitMask |= sieve_t(1) << bitIndex;

                index += prime;
                bitIndex += prime;
            }
            while (index <= lastIndex);

            sieve[wordIndex] &= ~bitMask;
        }
        else
        {
    #endif

            for (uint64_t index = firstUnmarked >> 1; index <= lastIndex; index += prime) 
                // Clear the bit in the word that corresponds to the last part of the index 
                sieve[WORD_INDEX(index)] &= ~(sieve_t(1) << BIT_INDEX(index));

    #if ROLLING_LIMIT > 0
        }
    #endif

    }
}

enum class Parallelization : char
{
    threads,
    blocks
};

template<typename E>
constexpr auto to_integral(E e) -> typename std::underlying_type<E>::type 
{
   return static_cast<typename std::underlying_type<E>::type>(e);
}

class Sieve 
{
    const uint64_t sieve_size;
    const uint64_t half_size;
    const uint32_t size_sqrt;
    const uint64_t buffer_word_size;
    const uint64_t buffer_byte_size;
    sieve_t *device_sieve_buffer;
    sieve_t *host_sieve_buffer;

    void unmark_multiples(Parallelization type, uint32_t primeCount, uint32_t *primeList) 
    {
        // Copy the first (square root of sieve size) buffer bytes to the device
        hipMemcpy(device_sieve_buffer, host_sieve_buffer, (size_sqrt >> 4) + 1, hipMemcpyHostToDevice);
        // Allocate device buffer for the list of primes and copy the prime list to it
        uint32_t *devicePrimeList;
        hipMalloc(&devicePrimeList, primeCount * sizeof(uint32_t));
        hipMemcpy(devicePrimeList, primeList, primeCount << 2, hipMemcpyHostToDevice);

        // Unmark multiples on the GPU and then release the prime list buffer
        switch(type)
        {
            case Parallelization::threads:
            {
                const uint32_t threadCount = min(MAX_THREADS, primeCount);
            #ifdef DEBUG
                printf("- starting thread multiple unmarking with %u threads.\n", threadCount);
            #endif
                unmark_multiples_threads<<<1, threadCount>>>(primeCount, devicePrimeList, half_size, size_sqrt, device_sieve_buffer);
            }
            break;

            case Parallelization::blocks:
            {
                const uint64_t sieveSpace = sieve_size - size_sqrt;
                uint64_t wordCount = sieveSpace << (WORD_SHIFT + 1);
                if (sieveSpace & SIEVE_WORD_MASK)
                    wordCount++;
                const uint32_t blockCount = (uint32_t)min(uint64_t(MAX_THREADS), wordCount);
                uint64_t blockSize = sieveSpace / blockCount;
                if (sieveSpace % blockCount)
                    blockSize++;

            #ifdef DEBUG
                printf("- starting block multiple unmarking with blockCount %u and blockSize %zu.\n", blockCount, blockSize);
            #endif
                unmark_multiples_blocks<<<blockCount, 1>>>(primeCount, devicePrimeList, half_size, size_sqrt, blockCount - 1, blockSize, device_sieve_buffer);
            }
            break;

            default:
                // This is some variation we don't know, so we warn and do nothing.
                fprintf(stderr, "WARNING: Parallelization type %d unknown, multiple unmarking skipped!\n\n", to_integral(type));
            break;
        }
        
        hipFree(devicePrimeList);

        // Copy the sieve buffer from the device to the host 
        hipMemcpy(host_sieve_buffer, device_sieve_buffer, buffer_byte_size, hipMemcpyDeviceToHost);
    #ifdef DEBUG
        printf("- device to host copy of sieve buffer complete.\n");
    #endif
    }

    public:

    Sieve(unsigned long size) :
        sieve_size(size),
        half_size(size >> 1),
        size_sqrt((uint32_t)sqrt(size) + 1),
        buffer_word_size((half_size >> WORD_SHIFT) + 1),
        buffer_byte_size(buffer_word_size * BYTES_PER_WORD)
    {
    #ifdef DEBUG
        printf("- constructing sieve with buffer_word_size %zu and buffer_byte_size %zu.\n", buffer_word_size, buffer_byte_size);
    #endif

        // Allocate and initialize device sieve buffer
        hipMalloc(&device_sieve_buffer, buffer_byte_size);

        const uint32_t blockCount = (uint32_t)min(uint64_t(MAX_THREADS), buffer_word_size);
        uint64_t blockSize = buffer_word_size / blockCount;
        if (buffer_word_size % blockCount)
            blockSize++;

    #ifdef DEBUG
        printf("- initializing device buffer with blockCount %u and blockSize %zu.\n", blockCount, blockSize);
    #endif
        initialize_buffer<<<blockCount, 1>>>(blockSize, buffer_word_size, device_sieve_buffer);

        // Allocate host sieve buffer and initialize the bytes up to the square root of the sieve size
        host_sieve_buffer = (sieve_t *)malloc(buffer_byte_size);
        memset(host_sieve_buffer, 255, (size_sqrt >> 4) + 1);
        hipDeviceSynchronize();
    #ifdef DEBUG
        printf("- post buffer initialization device sync complete.\n");
    #endif
    }

    ~Sieve() 
    {
        hipFree(device_sieve_buffer);
        free(host_sieve_buffer);
    }

    sieve_t *run(Parallelization type = Parallelization::threads)
    {
        // Calculate the size of the array we need to reserve for the primes we find up to and including the square root of
        //   the sieve size. x / (ln(x) - 1) is a good approximation, but often lower than the actual number, which would
        //   cause out-of-bound indexing. This is why we use x / (ln(x) - 1.2) to "responsibly over-allocate".
        const uint32_t primeListSize = uint32_t(double(size_sqrt) / (log(size_sqrt) - 1.2));

        uint32_t primeList[primeListSize];
        uint32_t primeCount = 0;

        // We clear multiples up to and including size_sqrt
        const uint32_t lastMultipleIndex = size_sqrt >> 1;

        for (uint32_t factor = 3; factor <= size_sqrt; factor += 2)
        {
            uint64_t index = factor >> 1;

            if (host_sieve_buffer[WORD_INDEX(index)] & (sieve_t(1) << BIT_INDEX(index))) 
            {
                primeList[primeCount++] = factor;

                for (index = (factor * factor) >> 1; index <= lastMultipleIndex; index += factor)
                    host_sieve_buffer[WORD_INDEX(index)] &= ~(sieve_t(1) << BIT_INDEX(index));
            }
        }

        unmark_multiples(type, primeCount, primeList);

        // Required to be truly compliant with Primes project rules
        return host_sieve_buffer;
    }

    uint64_t count_primes() 
    {
        uint64_t primeCount = 0;
        const uint64_t lastWord = WORD_INDEX(half_size);
        sieve_t word;

        for (uint64_t index = 0; index < lastWord; index++)
        {
            word = host_sieve_buffer[index];
            while (word) 
            {
                if (word & 1)
                    primeCount++;

                word >>= 1;
            }
        }

        word = host_sieve_buffer[lastWord];
        const uint32_t lastBit = BIT_INDEX(half_size);
        for (uint32_t index = 0; word && index <= lastBit; index++) 
        {
            if (word & 1)
                primeCount++;
            
            word >>= 1;
        }

        return primeCount;
    }
};

const std::map<uint64_t, const int> resultsDictionary =
{
    {             10UL, 4         }, // Historical data for validating our results - the number of primes
    {            100UL, 25        }, //   to be found under some limit, such as 168 primes under 1000
    {          1'000UL, 168       },
    {         10'000UL, 1229      },
    {        100'000UL, 9592      },
    {      1'000'000UL, 78498     },
    {     10'000'000UL, 664579    },
    {    100'000'000UL, 5761455   },
    {  1'000'000'000UL, 50847534  },
    { 10'000'000'000UL, 455052511 },
};

const std::map<Parallelization, const char *> parallelizationDictionary = 
{
    { Parallelization::threads, "threads" },
    { Parallelization::blocks,  "blocks"  }
};

// Assumes any first argument is the desired sieve size. Defaults to DEFAULT_SIEVE_SIZE.
uint64_t determineSieveSize(int argc, char *argv[])
{
    if (argc < 2)
        return DEFAULT_SIEVE_SIZE;

    const uint64_t sieveSize = strtoul(argv[1], nullptr, 0);

    if (sieveSize == 0) 
        return DEFAULT_SIEVE_SIZE;

    if (resultsDictionary.find(sieveSize) == resultsDictionary.end())
        fprintf(stderr, "WARNING: Results cannot be validated for selected sieve size of %zu!\n\n", sieveSize);
    
    return sieveSize;
}

void printResults(Parallelization type, uint64_t sieveSize, uint64_t primeCount, double duration, uint64_t passes)
{
    const auto expectedCount = resultsDictionary.find(sieveSize);
    const auto countValidated = expectedCount != resultsDictionary.end() && expectedCount->second == primeCount;
    const auto parallelizationEntry = parallelizationDictionary.find(type);
    const char *parallelizationLabel = parallelizationEntry != parallelizationDictionary.end() ? parallelizationEntry->second : "unknown";

    fprintf(stderr, "Passes: %zu, Time: %lf, Avg: %lf, Word size: %d, Max GPU threads: %d, Type: %s, Limit: %zu, Count: %zu, Validated: %d\n", 
            passes,
            duration,
            duration / passes,
            BITS_PER_WORD,
            MAX_THREADS,
            parallelizationLabel,
            sieveSize,
            primeCount,
            countValidated);

    printf("rbergen_faithful_cuda_%s;%zu;%f;1;algorithm=base,faithful=yes,bits=1\n\n", parallelizationLabel, passes, duration);
}

int main(int argc, char *argv[])
{
    const uint64_t sieveSize = determineSieveSize(argc, argv);

    Parallelization types[] = { Parallelization::blocks, Parallelization::threads };

    for (auto &type : types)
    {
        uint64_t passes = 0;

        Sieve *sieve = nullptr;

        const auto startTime = steady_clock::now();
        duration<double, std::micro> runTime;

    #ifndef DEBUG
        do
        {
    #endif

            delete sieve;

            sieve = new Sieve(sieveSize);
            sieve->run(type);

            passes++;

            runTime = steady_clock::now() - startTime;

    #ifndef DEBUG
        }
        while (duration_cast<seconds>(runTime).count() < 5);
    #else
        printf("\n");
    #endif

        const size_t primeCount = sieve->count_primes();
        
        delete sieve;

        printResults(type, sieveSize, primeCount, duration_cast<microseconds>(runTime).count() / 1000000.0, passes); 
    }
}
